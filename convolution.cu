#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include "dnn.hpp"

#include "hip/hip_runtime_api.h"
#include "hip/hip_runtime.h"
#include "cuda_device_runtime_api.h"
#include ""

using namespace std;

//Define the parameters if not defined externally
#ifndef Sy
  #define Sy 1
  #define Sx 1
#endif

#ifndef Tnn
  //Tiling Sizes
  #define Tnn 32
  #define Tn  16
  #define Ti  16

  #define Ty  8
  #define Tx  8
#endif

#ifndef CONCURRENT
  #define CONCURRENT true
#endif

#ifndef Nb
  #define Nb 4
#endif

#define NYPAD (Ny+Ky)
#define NXPAD (Nx+Kx)

#define NYSCL (Ny/Sy)
#define NXSCL (Nx/Sx)


#ifndef NUM_THREADS_Y
  // conv1 is 4, conv2 is 2
  #define NUM_THREADS_Y 1
  #define NUM_BLOCKS_Y 8
  // #define NUM_BLOCKS_Y (Ny/NUM_THREADS_Y)

  // conv1 and conv2 are both 2
  #define NUM_THREADS_X 1
  #define NUM_BLOCKS_X 8
  // #define NUM_BLOCKS_X (Nx/NUM_THREADS_X)

  // conv1 is 4, conv2 is 8
  #define NUM_THREADS_Z 32

#endif

#define NUM_BLOCKS_Z (Nn/NUM_THREADS_Z)


#define SYNAPSE_SIZE (1L*Nb*Ky*Kx*Nn*Ni)


// #define THREADS_PER_BLOCK (NUM_THREADS_Y*NUM_THREADS_X*NUM_THREADS_Z)

#define Synapse(i, j, p, q) (synapse[(i)*Kx * Ni * Nn + (j)*Ni*Nn + (p)*Nn + (q)])
#define Neuron_i(i, j, p) (neuron_i[(i)*NXPAD*Ni + (j)*Ni + (p)])
#define Neuron_n(i, j, p) (neuron_n[(i)*NXSCL*Nn + (j)*Nn + (p)])

VTYPE (*synapse)[Nb][Ky][Kx][Ni][Nn];

VTYPE (*neuron_i)[Nb][NYPAD][NXPAD][Ni];
VTYPE (*neuron_n)[Nb][NYSCL][NXSCL][Nn];
VTYPE (*neuron_n2)[Nb][NYSCL][NXSCL][Nn];

void fill_convolution_shared_simple(VTYPE (&synapse)[Nb][Ky][Kx][Ni][Nn], 
                                    VTYPE (&neuron_i)[Nb][NYPAD][NXPAD][Ni]) {
  for(int bb = 0; bb < Nb; ++bb) {
    for(int yy = 0; yy < Ky; ++yy) {
      for(int xx = 0; xx < Kx; ++xx) {
        for(int ni = 0; ni < Ni; ++ni) {
          for(int nn = 0; nn < Nn; ++nn) {
            synapse[bb][yy][xx][ni][nn] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX) - 0.5f;
          } } } } }
  for(int bb = 0; bb < Nb; ++bb) {
    for(int yy = 0; yy < NYPAD; ++yy) {
      for(int xx = 0; xx < NXPAD; ++xx) {      
        for(int ni = 0; ni < Ni; ++ni) {
          neuron_i[bb][yy][xx][ni] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX) - 0.5f;
    }  }  }  }
}

__global__
void convolution_layer_blocked(
                              const VTYPE *synapse, 
                              const VTYPE *neuron_i, 
                              VTYPE *neuron_n) {

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int idy = blockIdx.y * blockDim.y + threadIdx.y;
  int idn = blockIdx.z * blockDim.z + threadIdx.z;

  const int ySize = Ny/(NUM_THREADS_Y*NUM_BLOCKS_Y); 
  const int xSize = Nx/(NUM_THREADS_X*NUM_BLOCKS_X);
  const int nSize = NUM_THREADS_Z;
  
  __shared__ VTYPE sum[NUM_THREADS_Z];

  for (int y = idx*ySize; y < (idx+1)*ySize; ++y) { // tiling for y;

    for (int x = idy*xSize; x < (idy+1)*xSize; ++x) { // tiling for x;
      int n = idn;
      sum[n % nSize]=0;
      // sliding window;
      for (int ky = 0; ky < Ky; ++ky)
        for (int kx = 0; kx < Kx; ++kx)
          for (int i = 0; i < Ni; ++i) {
            VTYPE sv = Synapse(ky, kx, i, n);
            VTYPE nv = Neuron_i(ky + y, kx + x, i);
            sum[n % nSize] += sv*nv;
          }
        Neuron_n(y, x, n) = sum[n% nSize] > 0 ? sum[n% nSize] : sum[n% nSize]/4;
    }
  }
}

void  convolution_layer(VTYPE (&synapse)[Ky][Kx][Ni][Nn], 
                               VTYPE (&neuron_i)[NYPAD][NXPAD][Ni], 
                               VTYPE (&neuron_n)[NYSCL][NXSCL][Nn]) {
  VTYPE sum[Nn]={0};

  // — Original code — (excluding nn, ii loops)
  int yout = 0;
  for (int y = 0; y < Ny; y += Sy) { // tiling for y;
    int xout = 0;
    for (int x = 0; x < Ny; x += Sx) { // tiling for x;
      for (int nn = 0; nn < Nn; nn += Tn) {
        for (int n = nn; n < nn + Tn; n++) {
          sum[n]=0;
        }

        // sliding window;
        for (int ky = 0; ky < Ky; ky++)
          for (int kx = 0; kx < Kx; kx++)
            for (int i = 0; i < Ni; i++)
              for (int n = nn; n < nn + Tn; n++) {
                VTYPE sv = synapse[ky][kx][i][n];
                VTYPE nv = neuron_i[ky + y][kx + x][i];
                sum[n]+=sv*nv;
              }
        for (int n = nn; n < nn + Tn; n++) {
          neuron_n[yout][xout][n] = transfer(sum[n]);
        }
      }
      xout++; 
    }
    yout++;
  }
}

int main(const int argc, const char** argv) {
  cout << "allocating memory\n";

  synapse   = (VTYPE (*)[Nb][Ky][Kx][Ni][Nn])  aligned_malloc(64,  SYNAPSE_SIZE*sizeof(VTYPE));
  neuron_i  = (VTYPE (*)[Nb][NYPAD][NXPAD][Ni])aligned_malloc(64,Nb*NYPAD*NXPAD*Ni*sizeof(VTYPE));
  neuron_n  = (VTYPE (*)[Nb][NYSCL][NXSCL][Nn])aligned_malloc(64,Nb*NYSCL*NXSCL*Nn*sizeof(VTYPE));
  neuron_n2 = (VTYPE (*)[Nb][NYSCL][NXSCL][Nn])aligned_malloc(64,Nb*NYSCL*NXSCL*Nn*sizeof(VTYPE));

  hipError_t err = hipSuccess;

  cout << "initializing arrays\n";

  fill_convolution_shared_simple(*synapse,*neuron_i);

  float* d_synapse = NULL;
  err = hipMalloc((void**)&d_synapse, Nb*Ky*Kx*Nn*Ni*sizeof(VTYPE));
  if (err != hipSuccess) {
    cerr << "failed in allocating device synapse" << endl;
    exit(1);
  }
  err = hipMemcpy(d_synapse, synapse, Nb*Ky*Kx*Nn*Ni*sizeof(VTYPE), hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    cerr << "failed in copying device synapse" << endl;
    exit(1);
  }

  float* d_neuron_i = NULL;
  err = hipMalloc((void**)&d_neuron_i, Nb*NYPAD*NXPAD*Nn*sizeof(VTYPE));
  if (err != hipSuccess) {
    cerr << "failed in allocating device neuron_i" << endl;
    exit(1);
  }
  err = hipMemcpy(d_neuron_i, neuron_i, Nb*NYPAD*NXPAD*Nn*sizeof(VTYPE), hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    cerr << "failed in copying device neuron_i" << endl;
    exit(1);
  }

  float* d_neuron_n = NULL;
  err = hipMalloc((void**)&d_neuron_n, Nb*NYSCL*NXSCL*Nn*sizeof(VTYPE));
  if (err != hipSuccess) {
    cerr << "failed in allocating device neuron_n" << endl;
    exit(1);
  }

  cout << "starting computation\n";

  //Simple Version
  //begin_roi();

  for (int i = 0; i < Nb; ++i) {
    convolution_layer(((*synapse)[i]),
                      ((*neuron_i)[i]),
                      ((*neuron_n)[i]));
    cout << "simple: " << i << "\n";
  }


  //end_roi(Convolution, 0);

  cout << "simple version complete!\n";  

  dim3 dimGrid(NUM_BLOCKS_Y, NUM_BLOCKS_X, NUM_BLOCKS_Z);
  dim3 dimThread(NUM_THREADS_Y, NUM_THREADS_X, NUM_THREADS_Z);

  //Blocked Version
  begin_roi();

  if (CONCURRENT) {
    for (int i = 0; i < Nb; ++i)
      convolution_layer_blocked<<<dimGrid, dimThread>>>(&(d_synapse[i*Ky*Kx*Nn*Ni]), 
                                                        &(d_neuron_i[i*NYPAD*NXPAD*Nn]), 
                                                        &(d_neuron_n[i*NYSCL*NXSCL*Nn]));
  }
  else {
    for (int i = 0; i < Nb; ++i) {
      convolution_layer_blocked<<<dimGrid, dimThread>>>(&(d_synapse[i*Ky*Kx*Nn*Ni]), 
                                                        &(d_neuron_i[i*NYPAD*NXPAD*Nn]), 
                                                        &(d_neuron_n[i*NYSCL*NXSCL*Nn]));
      hipDeviceSynchronize();
    }
  }

  hipDeviceSynchronize();
  end_roi(Convolution, 1);

  if (err != hipSuccess) {
    cout << "Failed to launch classifier_layer_blocked kernel" << endl;
    exit(1);
  }

  cout << "here\n";
  
  err = hipMemcpy(neuron_n2, d_neuron_n, Nb*NYSCL*NXSCL*Nn*sizeof(VTYPE), hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    cout << "Failed to copy d_neuron_n from device to host" << endl;
    cout << hipGetErrorString(err) << endl;
    exit(1);
  }

  cout << "blocked computation complete!\n";  

  compare((VTYPE*)*neuron_n,(VTYPE*)*neuron_n2, Nb*NYSCL*NXSCL*Nn, Convolution, 1);

  cout << "compare done" << endl;


  // Free device memory
  err = hipFree(d_synapse);
  if (err != hipSuccess) {
    cout << "Failed to free device d_synapse" << endl;
    exit(1);
  }

  err = hipFree(d_neuron_i);
  if (err != hipSuccess) {
    cout << "Failed to free device d_neuron_i" << endl;
    exit(1);
  }

  err = hipFree(d_neuron_n);
  if (err != hipSuccess) {
    cout << "Failed to free device d_neuron_n" << endl;
    exit(1);
  }

  cout << "done\n";
  return 0;
}


