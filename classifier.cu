#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <cstdlib>
#include "dnn.hpp"

#include "hip/hip_runtime.h"
// #include <hip/hip_runtime.h
#include "hip/hip_runtime_api.h"
#include "hip/hip_runtime.h"
#include "cuda_device_runtime_api.h"
#include ""

using namespace std;

#ifndef Nb
  #define Nb 10  // Number of batches
#endif

//Define the parameters if not defined externally
#ifndef Nn
  #define Nn 128  // Number of Output Layers
  #define Ni 224  // Number of Input  Layers
#endif

#ifndef Tii
  // Tiling Sizes
  #define Tnn 32  
  #define Tii 32
  //#define Tn 5
  //#define Ti 25
  #define Tn 16
  #define Ti 16
#endif

// #define NUM_THREADS Tii
// #define NUM_BLOCKS Tn

#define NUM_THREADS 32
// #define NUM_BLOCKS 256
#define NUM_BLOCKS (Nn/NUM_THREADS)

// Macros for accessing 1D arrays in classifier kernel
#define Synapse(n, i) synapse[(n)*Ni + (i)]
#define Neuron_i(i) neuron_i[i]
#define Neuron_n(n) neuron_n[n]

//Arrays:
VTYPE (*synapse)[Nb][Nn][Ni];

VTYPE (*neuron_i)[Nb][Ni];
VTYPE (*neuron_n)[Nb][Nn];
VTYPE (*neuron_n2)[Nb][Nn];


// VTYPE synapse[Nb][Nn][Ni] __attribute__((aligned(64)));
// VTYPE neuron_i[Nb][Ni] __attribute__((aligned(64)));
// VTYPE neuron_n[Nb][Nn] __attribute__((aligned(64))),    neuron_n2[Nb][Nn] __attribute__((aligned(64)));

void fill_classifier(VTYPE (&synapse)[Nb][Nn][Ni], VTYPE (&neuron_i)[Nb][Ni], 
    VTYPE (&neuron_n)[Nb][Nn],   VTYPE (&neuron_n2)[Nb][Nn]) {
  for(int b = 0; b < Nb; ++b) {
    for(int n = 0; n < Nn; ++n) {
      for(int i = 0; i < Ni; ++i) {
        synapse[b][n][i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX) - 0.5f;
      }
    }
  }
  for(int b = 0; b < Nb; ++b) {
    for(int i = 0; i < Ni; ++i) {
      neuron_i[b][i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX) - 0.5f;
    }
  }
  for(int b = 0; b < Nb; ++b) {
    for(int n = 0; n < Nn; ++n) {
      neuron_n[b][n] = 0; //i;
      neuron_n2[b][n] = 0; //i;
    }
  }
}

void classifier_layer(VTYPE (&synapse)[Nn][Ni], VTYPE (&neuron_i)[Ni], VTYPE (&neuron_n)[Nn]) {
  // int total_calc=0;
  for (int n = 0; n < Nn; n++) {
    VTYPE temp=0;
    for (int i = 0; i < Ni; i++) {
      temp += synapse[n][i] * neuron_i[i];
    }
    neuron_n[n] = transfer(temp);
  }
}

__global__
void classifier_layer_blocked(const VTYPE *synapse, const VTYPE *neuron_i, 
                              VTYPE *neuron_n) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
                          
  for (int n = idx*(Nn/(NUM_THREADS*NUM_BLOCKS)); n < (idx+1)*(Nn/(NUM_THREADS*NUM_BLOCKS)); ++n) {
    VTYPE temp_0=0;

    for (int i = 0; i < Ni; ++i) {
      // for (int ii = 0; ii < Ti; ++ii){
        temp_0 += Synapse(n, i) * neuron_i[i];
      // }
    }
    neuron_n[n] = temp_0 > 0 ? temp_0 : temp_0/4;
  }
}

int main(int argc, char** argv) {

  synapse   = (VTYPE (*)[Nb][Nn][Ni]) aligned_malloc(64,Nb*Nn*Ni*sizeof(VTYPE));
  neuron_i  = (VTYPE (*)[Nb][Ni]) aligned_malloc(64,Nb*Ni*sizeof(VTYPE));
  neuron_n  = (VTYPE (*)[Nb][Nn]) aligned_malloc(64,Nb*Nn*sizeof(VTYPE));
  neuron_n2 = (VTYPE (*)[Nb][Nn]) aligned_malloc(64,Nb*Nn*sizeof(VTYPE));


  // Error code to check return values for CUDA calls
  hipError_t err = hipSuccess;


  // Initialize arrays for run
  cout << "initializing arrays\n";
  fill_classifier(*synapse,*neuron_i,*neuron_n,*neuron_n2);


  // Allocate and copy to Device arrays
  float* d_synapse = NULL;
  err = hipMalloc((void**)&d_synapse, Nb*Nn*Ni*sizeof(VTYPE));
  if (err != hipSuccess) {
    cerr << "failed in allocating device synapse" << endl;
    exit(1);
  }
  err = hipMemcpy(d_synapse, synapse, Nb*Nn*Ni*sizeof(VTYPE), hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    cerr << "failed in copying device synapse" << endl;
    exit(1);
  }

  float* d_neuron_i = NULL;
  err = hipMalloc((void**)&d_neuron_i, Nb*Ni*sizeof(VTYPE));
  if (err != hipSuccess) {
    cerr << "failed in allocating device neuron_i" << endl;
    exit(1);
  }
  err = hipMemcpy(d_neuron_i, neuron_i, Nb*Ni*sizeof(VTYPE), hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    cerr << "failed in copying device neuron_i" << endl;
    exit(1);
  }

  float* d_neuron_n = NULL;
  err = hipMalloc((void**)&d_neuron_n, Nb*Nn*sizeof(VTYPE));
  if (err != hipSuccess) {
    cerr << "failed in allocating device neuron_n" << endl;
    exit(1);
  }
  cout << "starting computation\n";

  // Perform and time simple run
  // begin_roi();
  // for (int i = 0; i < Nb; i++) {
  //   classifier_layer(synapse[i],neuron_i[i],neuron_n[i]);
  // }
  // end_roi(Classifier, 0);

  cout << "simple version complete!\n";

  // Create Stream Objects for concurrent execution
  int nstreams = Nb;
  // allocate and initialize an array of stream handles
  hipStream_t *streams = (hipStream_t *) malloc(nstreams * sizeof(hipStream_t));
  for (int i = 0; i < nstreams; i++)
  {
    checkCudaErrors(hipStreamCreate(&(streams[i])));
  }

  // randomize the order of the batches
  int order[Nb];
  for (int i = 0; i < Nb; ++i) {
    order[i] = i;
  }
  for (int i=0; i<Nb; i++) {
    int r = rand() % Nb;
    int temp = order[i];
    order[i] = order[r];
    order[r] = temp;
  }
  for (int i = 0; i < Nb; ++i) {
    cout << order[i] << " ";
  }
  cout << "\n";

  // Perform and time the blocked, distributed run
  dim3 dimGrid(NUM_BLOCKS, 1, 1);
  dim3 dimThread(NUM_THREADS, 1, 1);

  begin_roi();
  // classifier_layer_blocked(synapse,neuron_i,neuron_n2);
  // classifier_layer_blocked<<<dimGrid, dimThread>>>(d_synapse, d_neuron_i, d_neuron_n);

  if (!CONCURRENT) {
    for (int i = 0; i < Nb; ++i) {
      classifier_layer_blocked<<<dimGrid, dimThread>>>(&(d_synapse[order[i]*Nn*Ni]), 
                                                        &(d_neuron_i[order[i]*Ni]), 
                                                        &(d_neuron_n[order[i]*Nn]));
      hipDeviceSynchronize();
    }
    cout << "seq\n";
  }
  else {
    for (int i = 0; i < Nb; i++) {
      classifier_layer_blocked<<<dimGrid, dimThread, 0, streams[i]>>>(&(d_synapse[order[i]*Nn*Ni]), 
                                                        &(d_neuron_i[order[i]*Ni]), 
                                                        &(d_neuron_n[order[i]*Nn]));
    }
    cout << "con\n";
  }

  hipDeviceSynchronize();
  end_roi(Classifier, 1);
  
  err = hipGetLastError();
  if (err != hipSuccess) {
    cout << "Failed to launch classifier_layer_blocked kernel" << endl;
    exit(1);
  }
  cout << "blocked computation complete!\n"; 
  
  err = hipMemcpy(neuron_n2, d_neuron_n, Nb*Nn*sizeof(VTYPE), hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    cout << "Failed to copy d_neuron_n from device to host" << endl;
    cout << hipGetErrorString(err) << endl;
    exit(1);
  }

  // Compare results
  // compare(&neuron_n[0][0],&neuron_n2[0][0], Nb*Nn, Classifier, 1);
  compare2(Classifier, 1);
  cout << "compare done" << endl;


  // Free device memory
  err = hipFree(d_synapse);
  if (err != hipSuccess) {
    cout << "Failed to free device d_synapse" << endl;
    exit(1);
  }

  err = hipFree(d_neuron_i);
  if (err != hipSuccess) {
    cout << "Failed to free device d_neuron_i" << endl;
    exit(1);
  }

  err = hipFree(d_neuron_n);
  if (err != hipSuccess) {
    cout << "Failed to free device d_neuron_n" << endl;
    exit(1);
  }

  cout << "Done!" << endl;
  return 0;
}

